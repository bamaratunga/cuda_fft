#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
#include<hipfft/hipfftXt.h>
#include<hip/hip_complex.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Write ouput to CSV file
__host__ void writeCSV(double * input, int idx, unsigned int N){
    char fname[0x100];
    snprintf(fname, sizeof(fname), "output_%d.csv", idx);
    FILE *fp = fopen(fname, "w");

    for(int col = 0; col < N; col++){
        for(int row = 0; row < N-1; row++){
            fprintf(fp, "%lf, ", input[row + N * col]);
        }
        fprintf(fp, "%lf", input[N-1 + N * col]);
        fprintf(fp, "\n");
    }
    fclose(fp);
}

// forward FFT (inplace)
// real data are put in contiguous data array, input[1:Nx, 1:Ny]
// but size of input is bigger, say  Nx * 2*(Ny>>1 +1) doublereal

// output:
// input is a complex array with size Nx*(Ny>>1 + 1)

void  fft2(hipDoubleComplex * inData, const unsigned int N) {

	hipfftDoubleComplex *d_inData = NULL;

	gpuErrChk(hipMalloc(&d_inData, N * N * sizeof(hipfftDoubleComplex)));

    gpuErrChk(hipMemcpy(d_inData, inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

	hipfftHandle plan;
    hipfftResult flag;

	flag = hipfftPlan2d(&plan, N, N, HIPFFT_Z2Z);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftPlan2d fails!\n");

	flag = hipfftExecZ2Z(plan, d_inData, d_inData, HIPFFT_FORWARD);
	if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftExecR2C fails!\n");

    gpuErrChk(hipDeviceSynchronize());
	gpuErrChk(hipMemcpy(inData, d_inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );

	flag = hipfftDestroy(plan);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftDestroy fails!\n");
	gpuErrChk(hipFree(d_inData));
}


int main(int argc, char** argv){

  if(argc < 2) {
      printf("Enter the dimension size as argument!\n");
      exit(EXIT_FAILURE);
  }

  int N = atoi(argv[1]);

  // Complex data input
  hipDoubleComplex * inputData = (hipDoubleComplex *)malloc(N * N * sizeof(hipDoubleComplex));
  // Real data
  double * outputData = (double *)malloc(N * N * sizeof(double));

  int slit_height = 1;
  int slit_width  = 2;
  int slit_dist   = 7;

  // TODO: Create this data on the device itself
  // Create double slit
  for (int j = 0; j < N; j++){
    for (int i = 0; i < N; i++){
      inputData[j * N + i] = make_hipDoubleComplex(0.0, 0.0);
      // Set slit positions to 1
      if ((abs(i-N/2) <= slit_dist+slit_width) && (abs(i-N/2) >= slit_dist) && (abs(j-N/2) <= slit_height)){
        inputData[j * N + i] = make_hipDoubleComplex(1.0, 0.0);
      } // printf("%0.0lf ", reInput[j * N + i]);
    } // printf("\n");
  }

  printf("Running fft for %d x %d = %d = 2 ^ %d data points...\n", N, N, N*N, (int)(log(N*N)/log(2)));

  clock_t start, end;
  double cpu_time_used;

  start = clock();
  fft2(inputData, N);
  end = clock();

  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

  printf("Runtime = %lfs\n", cpu_time_used);

  for(int i = 0; i < N*N; i++){
      outputData[i] = hipCreal(inputData[i]) * hipCreal(inputData[i])
                    + hipCimag(inputData[i]) * hipCimag(inputData[i]);
  }

  printf("Writing output data...\n");
  writeCSV(outputData, 0, N);

  free(inputData);
  free(outputData);

  return 0;
}

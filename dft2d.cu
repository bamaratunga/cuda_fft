#include "hip/hip_runtime.h"
/***********************************************************
*
* Developed for Seminar in Parallelisation of Physics
* Calculations on GPUs with CUDA, Department of Physics
* Technical University of Munich.
*
* Author: Binu Amaratunga
*
*
***********************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>
#include <hip/hip_runtime.h>

#include "controls.h"
#include "utils.h"

__global__ void fft(double * inputData, double * amplitudeOut, int N){
    double realOut = 0;
    double imagOut = 0;

    int xWave = blockIdx.x*blockDim.x + threadIdx.x;
    int yWave = blockIdx.y*blockDim.y + threadIdx.y;

    int height = N;
    int width = N;

    for (int ySpace = 0; ySpace < height; ySpace++) {
        for (int xSpace = 0; xSpace < width; xSpace++) {
            // Compute real, imag, and ampltude.
            realOut += (inputData[ySpace * width + xSpace] * cos(2.0 * M_PI * ((1.0 * xWave * xSpace / width) + (1.0 * yWave * ySpace / height))));
            imagOut -= (inputData[ySpace * width + xSpace] * sin(2.0 * M_PI * ((1.0 * xWave * xSpace / width) + (1.0 * yWave * ySpace / height))));
        }
    }
    // amplitudeOut[yWave * n + xWave] = sqrt(realOut * realOut + imagOut * imagOut);
    amplitudeOut[yWave * N + xWave] = (realOut * realOut + imagOut * imagOut);
}


int main(int argc, char **argv) {

    if(argc < 2) {
        printf("Enter the dimension size as argument!\n");
        exit(EXIT_FAILURE);
    }

    int N = atoi(argv[1]);

    int i, j;

    double * inputData = (double *)malloc(N * N * sizeof(double));
    double * amplitudeOut = (double *)malloc(N * N * sizeof(double));

    // TODO: Create this data on the device itself
    for (j = 0; j < N; j++){
        for (i = 0; i < N; i++){
            inputData[j*N + i] = 0.0;
            // Set slit positions to 1
            if ((abs(i-N/2) <= 10) && (abs(i-N/2) >= 8) && (abs(j-N/2) <= 4)){
              inputData[j*N + i] = 1.0;
            }
            amplitudeOut[j*N + i] = 0.0;
        }
    }

    clock_t start, end;
    double cpu_time_used;

    printf("Running fft for %d x %d = %d = 2 ^ %d data points...\n", N, N, N*N, (int)(log(N*N)/log(2)));

    start = clock();

    double * d_inputData = NULL;
    double * d_amplitudeOut = NULL;

    gpuErrChk(hipMalloc(&d_inputData, N * N * sizeof(double)));
    gpuErrChk(hipMalloc(&d_amplitudeOut, N * N * sizeof(double)));

    gpuErrChk(hipMemcpy(d_inputData, inputData, N * N * sizeof(double), hipMemcpyHostToDevice));

    dim3 gridSize(N / 32, N / 32);
    dim3 blockSize( 32, 32); // Multiples of 32

    fft<<<gridSize, blockSize>>>(d_inputData, d_amplitudeOut, N);

    gpuErrChk(hipDeviceSynchronize());

    gpuErrChk(hipMemcpy(amplitudeOut, d_amplitudeOut, N * N * sizeof(double), hipMemcpyDeviceToHost));

    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Runtime = %lfs\n", cpu_time_used);

    printf("Writing output data...\n");
    writeCSV(amplitudeOut, 0, N);

    gpuErrChk(hipFree(d_inputData));
    gpuErrChk(hipFree(d_amplitudeOut));

    free(inputData);
    free(amplitudeOut);

    return 0;
}

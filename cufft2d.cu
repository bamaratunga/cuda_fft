#include "hip/hip_runtime.h"
/***********************************************************
*
* Developed for Seminar in Parallelisation of Physics
* Calculations on GPUs with CUDA, Department of Physics
* Technical University of Munich.
*
* Author: Binu Amaratunga
*
*
***********************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>

#include "controls.h"
#include "utils.h"

/*************************************
* Compute 2D FFT with cuFFT
* output
*
*
*
**************************************/

void  fft2(hipDoubleComplex * inData, const unsigned int N) {

	hipfftDoubleComplex *d_inData = NULL;

	gpuErrChk(hipMalloc(&d_inData, N * N * sizeof(hipfftDoubleComplex)));

    gpuErrChk(hipMemcpy(d_inData, inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

	hipfftHandle plan;
    hipfftResult flag;

	flag = hipfftPlan2d(&plan, N, N, HIPFFT_Z2Z);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftPlan2d fails!\n");

	flag = hipfftExecZ2Z(plan, d_inData, d_inData, HIPFFT_FORWARD);
	if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftExecR2C fails!\n");

    gpuErrChk(hipDeviceSynchronize());
	gpuErrChk(hipMemcpy(inData, d_inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );

	flag = hipfftDestroy(plan);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftDestroy fails!\n");
	gpuErrChk(hipFree(d_inData));
}


int main(int argc, char** argv){

  if(argc < 2) {
      printf("Enter the dimension size as argument!\n");
      exit(EXIT_FAILURE);
  }

  int N = atoi(argv[1]);

  // Complex data input
  hipDoubleComplex * inputData = (hipDoubleComplex *)malloc(N * N * sizeof(hipDoubleComplex));
  // Real data
  double * outputData = (double *)malloc(N * N * sizeof(double));

  int slit_height = 1;
  int slit_width  = 2;
  int slit_dist   = 7;

  // TODO: Create this data on the device itself
  // Create double slit
  for (int j = 0; j < N; j++){
    for (int i = 0; i < N; i++){
      inputData[j * N + i] = make_hipDoubleComplex(0.0, 0.0);
      // Set slit positions to 1
      if ((abs(i-N/2) <= slit_dist+slit_width) && (abs(i-N/2) >= slit_dist) && (abs(j-N/2) <= slit_height)){
        inputData[j * N + i] = make_hipDoubleComplex(1.0, 0.0);
      } // printf("%0.0lf ", reInput[j * N + i]);
    } // printf("\n");
  }

  printf("Running fft for %d x %d = %d = 2 ^ %d data points...\n", N, N, N*N, (int)(log(N*N)/log(2)));

  clock_t start, end;
  double cpu_time_used;

  start = clock();
  fft2(inputData, N);
  // TODO: Do this in cuBLAS
  for(int i = 0; i < N*N; i++){
      outputData[i] = hipCreal(inputData[i]) * hipCreal(inputData[i])
                    + hipCimag(inputData[i]) * hipCimag(inputData[i]);
  }
  end = clock();

  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

  printf("Runtime = %lfs\n", cpu_time_used);

  printf("Writing output data...\n");
  writeCSV(outputData, 0, N);

  free(inputData);
  free(outputData);

  return 0;
}

#include<stdio.h>
#include<stdlib.h>

#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
#include<hipfft/hipfftXt.h>
#include<hip/hip_complex.h>

const int N = 4096;

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Write ouput to CSV file
__host__ void writeCSV(double * input, int idx){
    char fname[0x100];
    snprintf(fname, sizeof(fname), "output_%d.csv", idx);
    FILE *fp = fopen(fname, "w");

    for(int col = 0; col < N; col++){
        for(int row = 0; row < N-1; row++){
            fprintf(fp, "%lf, ", input[row + N * col]);
        }
        fprintf(fp, "%lf", input[N-1 + N * col]);
        fprintf(fp, "\n");
    }
    fclose(fp);
}

// forward FFT (inplace)
// real data are put in contiguous data array, input[1:Nx, 1:Ny]
// but size of input is bigger, say  Nx * 2*(Ny>>1 +1) doublereal

// output:
// input is a complex array with size Nx*(Ny>>1 + 1)

void  fft2(hipDoubleComplex * inData, const unsigned int N) {

	hipfftDoubleComplex *d_inData = NULL;

	gpuErrChk(hipMalloc(&d_inData, N * N * sizeof(hipfftDoubleComplex)));

    gpuErrChk(hipMemcpy(d_inData, inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

	hipfftHandle plan;
    hipfftResult flag;

	flag = hipfftPlan2d(&plan, N, N, HIPFFT_Z2Z);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftPlan2d fails!\n");

	flag = hipfftExecZ2Z(plan, d_inData, d_inData, HIPFFT_FORWARD);
	if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftExecR2C fails!\n");

    gpuErrChk(hipDeviceSynchronize());
	gpuErrChk(hipMemcpy(inData, d_inData, N * N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost) );

	flag = hipfftDestroy(plan);
    if ( HIPFFT_SUCCESS != flag ) printf("2D: hipfftDestroy fails!\n");
	gpuErrChk(hipFree(d_inData));
}


int main()
{
  // Complex data input
  hipDoubleComplex * inputData = (hipDoubleComplex *)malloc(N * N * sizeof(hipDoubleComplex));
  // Real data
  double * outputData = (double *)malloc(N * N * sizeof(double));

  int slit_height = 4;
  int slit_width  = 2;
  int slit_dist   = 8;

  // TODO: Create this data on the device itself
  // Create double slit
  for (int j = 0; j < N; j++){
    for (int i = 0; i < N; i++){
      inputData[j * N + i] = make_hipDoubleComplex(0.0, 0.0);
      // Set slit positions to 1
      if ((abs(i-N/2) <= slit_dist+slit_width) && (abs(i-N/2) >= slit_dist) && (abs(j-N/2) <= slit_height)){
        inputData[j * N + i] = make_hipDoubleComplex(1.0, 0.0);
      } // printf("%0.0lf ", reInput[j * N + i]);
    } // printf("\n");
  }

  fft2(inputData, N);

  for(int i = 0; i < N*N; i++){
      outputData[i] = hipCreal(inputData[i]) * hipCreal(inputData[i])
                    + hipCimag(inputData[i]) * hipCimag(inputData[i]);
  }

  writeCSV(outputData, 0);

  free(inputData);
  free(outputData);

  return 0;
}
